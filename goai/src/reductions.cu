#include "hip/hip_runtime.h"
#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>

#include <cub/block/block_reduce.cuh>

#include <limits>

#define REDUCTION_BLOCK_SIZE 128


struct IdentityLoader{
    template<typename T>
    __device__
    T operator() (const T *ptr, int pos) const {
        return ptr[pos];
    }
};

/*
Generic reduction implementation with support for validity mask
*/

template<typename T, typename F, typename Ld>
__global__
void gpu_reduction_op(const T *data, const gdf_valid_type *mask,
                      gdf_size_type size, T *results, F functor, T identity,
                      Ld loader)
{
    typedef hipcub::BlockReduce<T, REDUCTION_BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;

    int step = blksz * gridsz;

    T agg = identity;

    for (int base=blkid * blksz; base<size; base+=step) {
        // Threadblock synchronous loop
        int i = base + tid;
        // load
        T loaded = identity;
        if (i < size)
            loaded = loader(data, i);
        // set invalid location to identity
        if ( !gdf_is_valid(mask, i) ) {
             loaded = identity;
        }
        // Block reduce
        T temp = BlockReduce(temp_storage).Reduce(loaded, functor);
        // Add current block
        agg = functor(agg, temp);
    }
    // First thread of each block stores the result.
    if (tid == 0)
        results[blkid] = agg;
}



template<typename T, typename F>
struct ReduceOp {
    static
    gdf_error launch(gdf_column *input, T identity, T *output,
                     gdf_size_type output_size) {

        // 1st round
        //    Partially reduce the input into *output_size* length.
        //    Each block computes one output in *output*.
        //    output_size == gridsize
        typedef typename F::Loader Ld1;
        F functor1;
        Ld1 loader1;
        launch_once((const T*)input->data, input->valid, input->size,
                    (T*)output, output_size, identity, functor1, loader1);
        CUDA_CHECK_LAST();

        // 2nd round
        //    Finish the partial reduction (if needed).
        //    A single block reduction that computes one output stored to the
        //    first index in *output*.
        if ( output_size > 1 ) {
            typedef typename F::second F2;
            typedef typename F2::Loader Ld2;
            F2 functor2;
            Ld2 loader2;

            launch_once(output, nullptr, output_size,
                        output, 1, identity, functor2, loader2);
            CUDA_CHECK_LAST();
        }

        return GDF_SUCCESS;
    }

    template <typename Functor, typename Loader>
    static
    void launch_once(const T *data, gdf_valid_type *valid, gdf_size_type size,
                     T *output, gdf_size_type output_size, T identity,
                     Functor functor, Loader loader) {
        // find needed gridsize
        // use atmost REDUCTION_BLOCK_SIZE blocks
        int blocksize = REDUCTION_BLOCK_SIZE;
        int gridsize = (output_size < REDUCTION_BLOCK_SIZE?
                        output_size : REDUCTION_BLOCK_SIZE);

        // launch kernel
        gpu_reduction_op<<<gridsize, blocksize>>>(
            // inputs
            data, valid, size,
            // output
            output,
            // action
            functor,
            // identity
            identity,
            // loader
            loader
        );
    }

};


struct DeviceSum {
    typedef IdentityLoader Loader;
    typedef DeviceSum second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs + rhs;
    }
};

struct DeviceProduct {
    typedef IdentityLoader Loader;
    typedef DeviceProduct second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs * rhs;
    }
};


struct DeviceSumSquared {
    struct Loader {
        template <typename T>
        __device__
        T operator() (const T* ptr, int pos) const {
            T val = ptr[pos];   // load
            return val * val;   // squared
        }
    };
    // round 2 just uses the basic sum reduction
    typedef DeviceSum second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) const {
        return lhs + rhs;
    }
};


struct DeviceMin {
    typedef IdentityLoader Loader;
    typedef DeviceMin second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs <= rhs? lhs: rhs;
    }
};


struct DeviceMax {
    typedef IdentityLoader Loader;
    typedef DeviceMax second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs >= rhs? lhs: rhs;
    }
};

#define DEF_REDUCE_OP_NUM(F)                                                      \
gdf_error F##_generic(gdf_column *col, void *dev_result,                          \
                          gdf_size_type dev_result_size) {                        \
    switch ( col->dtype ) {                                                       \
    case GDF_FLOAT64: return F##_f64(col, (double*)dev_result, dev_result_size);  \
    case GDF_FLOAT32: return F##_f32(col, (float*)dev_result, dev_result_size);   \
    case GDF_INT64:   return F##_i64(col, (int64_t*)dev_result, dev_result_size); \
    case GDF_INT32:   return F##_i32(col, (int32_t*)dev_result, dev_result_size); \
    case GDF_INT8:    return F##_i8(col,  (int8_t*)dev_result, dev_result_size);  \
    default:          return GDF_UNSUPPORTED_DTYPE;                               \
    }                                                                             \
}

#define DEF_REDUCE_OP_REAL(F)                                                     \
gdf_error F##_generic(gdf_column *col, void *dev_result,                          \
                          gdf_size_type dev_result_size) {                        \
    switch ( col->dtype ) {                                                       \
    case GDF_FLOAT64: return F##_f64(col, (double*)dev_result, dev_result_size);  \
    case GDF_FLOAT32: return F##_f32(col, (float*)dev_result, dev_result_size);   \
    default:          return GDF_UNSUPPORTED_DTYPE;                               \
    }                                                                             \
}

#define DEF_REDUCE_IMPL(F, OP, T, ID)                                         \
gdf_error F(gdf_column *col, T *dev_result, gdf_size_type dev_result_size) {  \
    return ReduceOp<T, OP>::launch(col, ID, dev_result, dev_result_size);     \
}


unsigned int gdf_reduce_optimal_output_size() {
    return REDUCTION_BLOCK_SIZE;
}


/* Sum */

DEF_REDUCE_OP_NUM(gdf_sum)
DEF_REDUCE_IMPL(gdf_sum_f64, DeviceSum, double, 0)
DEF_REDUCE_IMPL(gdf_sum_f32, DeviceSum, float, 0)
DEF_REDUCE_IMPL(gdf_sum_i64, DeviceSum, int64_t, 0)
DEF_REDUCE_IMPL(gdf_sum_i32, DeviceSum, int32_t, 0)
DEF_REDUCE_IMPL(gdf_sum_i8,  DeviceSum, int8_t, 0)

/* Product */

DEF_REDUCE_OP_NUM(gdf_product)
DEF_REDUCE_IMPL(gdf_product_f64, DeviceProduct, double, 1)
DEF_REDUCE_IMPL(gdf_product_f32, DeviceProduct, float, 1)
DEF_REDUCE_IMPL(gdf_product_i64, DeviceProduct, int64_t, 1)
DEF_REDUCE_IMPL(gdf_product_i32, DeviceProduct, int32_t, 1)
DEF_REDUCE_IMPL(gdf_product_i8,  DeviceProduct, int8_t, 1)

/* Sum Squared */

DEF_REDUCE_OP_REAL(gdf_sum_squared)
DEF_REDUCE_IMPL(gdf_sum_squared_f64, DeviceSumSquared, double, 0)
DEF_REDUCE_IMPL(gdf_sum_squared_f32, DeviceSumSquared, float, 0)

/* Min */

DEF_REDUCE_OP_NUM(gdf_min)
DEF_REDUCE_IMPL(gdf_min_f64, DeviceMin, double, std::numeric_limits<double>::max())
DEF_REDUCE_IMPL(gdf_min_f32, DeviceMin, float, std::numeric_limits<float>::max())
DEF_REDUCE_IMPL(gdf_min_i64, DeviceMin, int64_t, std::numeric_limits<int64_t>::max())
DEF_REDUCE_IMPL(gdf_min_i32, DeviceMin, int32_t, std::numeric_limits<int32_t>::max())
DEF_REDUCE_IMPL(gdf_min_i8, DeviceMin, int8_t, std::numeric_limits<int8_t>::max())

/* Max */

DEF_REDUCE_OP_NUM(gdf_max)
DEF_REDUCE_IMPL(gdf_max_f64, DeviceMax, double, std::numeric_limits<double>::lowest())
DEF_REDUCE_IMPL(gdf_max_f32, DeviceMax, float, std::numeric_limits<float>::lowest())
DEF_REDUCE_IMPL(gdf_max_i64, DeviceMax, int64_t, std::numeric_limits<int64_t>::lowest())
DEF_REDUCE_IMPL(gdf_max_i32, DeviceMax, int32_t, std::numeric_limits<int32_t>::lowest())
DEF_REDUCE_IMPL(gdf_max_i8, DeviceMax, int8_t,  std::numeric_limits<int8_t>::lowest())
