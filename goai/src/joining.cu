/*

Adapted from https://github.com/moderngpu/moderngpu which has the following license:

> Copyright (c) 2016, Sean Baxter
> All rights reserved.
>
> Redistribution and use in source and binary forms, with or without
> modification, are permitted provided that the following conditions are met:
>
> 1. Redistributions of source code must retain the above copyright notice, this
>    list of conditions and the following disclaimer.
> 2. Redistributions in binary form must reproduce the above copyright notice,
>    this list of conditions and the following disclaimer in the documentation
>    and/or other materials provided with the distribution.
>
> THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
> ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
> WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
> DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
> ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
> (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
> LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
> ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
> (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
> SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
>
> The views and conclusions contained in the software and documentation are those
> of the authors and should not be interpreted as representing official policies,
> either expressed or implied, of the FreeBSD Project.
*/


#include <gdf/gdf.h>
#include <gdf/errorutils.h>


#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>


#include <memory>
#include <iostream>

namespace {

using namespace mgpu;

template <typename T>
void dump_mem(const char name[], const mem_t<T> & mem) {

    auto data = from_mem(mem);
    std::cout << name << " = " ;
    for (int i=0; i < data.size(); ++i) {
        std::cout << data[i] << ", ";
    }
    std::cout << "\n";
}


struct _join_bounds {
    mem_t<int> lower, upper;
};


template<typename launch_arg_t = empty_t,
  typename a_it, typename b_it, typename comp_t>
_join_bounds compute_join_bounds(a_it a, int a_count, b_it b, int b_count,
    comp_t comp, context_t& context) {

    mem_t<int> lower(a_count, context);
    mem_t<int> upper(a_count, context);
    sorted_search<bounds_lower, launch_arg_t>(a, a_count, b, b_count,
    lower.data(), comp, context);
    sorted_search<bounds_upper, launch_arg_t>(a, a_count, b, b_count,
    upper.data(), comp, context);

    // Prepare output
    _join_bounds bounds;
    lower.swap(bounds.lower);
    upper.swap(bounds.upper);
    return bounds;
}


mem_t<int> scan_join_bounds(const _join_bounds &bounds, int a_count, int b_count,
                            context_t &context, bool isInner,
                            int &out_join_count)
{
    // Compute output ranges by scanning upper - lower. Retrieve the reduction
    // of the scan, which specifies the size of the output array to allocate.
    mem_t<int> scanned_sizes(a_count, context);
    const int* lower_data = bounds.lower.data();
    const int* upper_data = bounds.upper.data();

    mem_t<int> count(1, context);

    if (isInner){
        transform_scan<int>([=]MGPU_DEVICE(int index) {
            return upper_data[index] - lower_data[index];
        }, a_count, scanned_sizes.data(), plus_t<int>(), count.data(), context);
    } else {
        transform_scan<int>([=]MGPU_DEVICE(int index) {
            auto out = upper_data[index] - lower_data[index];
            if ( upper_data[index] == lower_data[index] ){
                // for left-only keys, allocate a slot
                out += 1;
            }
            return out;
        }, a_count, scanned_sizes.data(), plus_t<int>(), count.data(), context);
    }

    // Prepare output
    out_join_count = from_mem(count)[0];
    return scanned_sizes;
}


template<typename launch_arg_t = empty_t>
mem_t<int> compute_joined_indices(const _join_bounds &bounds,
                                   const mem_t<int> &scanned_sizes,
                                   int a_count, int join_count,
                                   context_t &context,
                                   bool isInner, int append_count=0)
{
    // Allocate an int output array and use load-balancing search to compute
    // the join.

    const int* lower_data = bounds.lower.data();
    const int* upper_data = bounds.upper.data();

    // for outer join: allocate extra space for appending the right indices
    int output_npairs = join_count + append_count;
    mem_t<int> output(2 * output_npairs, context);
    int* output_data = output.data();

    if (isInner){
        // Use load-balancing search on the segments. The output is a pair with
        // a_index = seg and b_index = lower_data[seg] + rank.
        auto k = [=]MGPU_DEVICE(int index, int seg, int rank, const int *lower) {
            output_data[index] = seg;
            output_data[index + output_npairs] = lower[seg] + rank;
        };

        transform_lbs<launch_arg_t>(k, join_count, scanned_sizes.data(), a_count,
                                    context, lower_data);
    } else {
        // Use load-balancing search on the segments. The output is a pair with
        // a_index = seg
        // b_index = lower_data[seg] + rank { if lower_data[seg] != upper_data[seg] }
        //         = -1                     { otherwise }
        auto k = [=]MGPU_DEVICE(int index, int seg, int rank, tuple<int, int> lower_upper) {
            auto lower = get<0>(lower_upper);
            auto upper = get<1>(lower_upper);
            auto result = lower + rank;
            if ( lower == upper ) result = -1;
            output_data[index] = seg;
            output_data[index + output_npairs] = result;
        };
        transform_lbs<launch_arg_t>(k, join_count, scanned_sizes.data(), a_count,
                                    make_tuple(lower_data, upper_data), context);
    }
    return output;
}


template<typename launch_arg_t = empty_t, typename T>
void outer_join_append_right(T *output_data,
                             const mem_t<int> &matches,
                             int append_count, int join_count,
                             context_t &context) {
    int output_npairs = join_count + append_count;
    auto appender = [=]MGPU_DEVICE(int index, int seg, int rank) {
        output_data[index + join_count] = -1;
        output_data[index + join_count + output_npairs] = seg;
    };
    transform_lbs<launch_arg_t>(appender, append_count, matches.data(),
                                matches.size(), context);
}

template<typename launch_arg_t = empty_t,
         typename a_it, typename b_it, typename comp_t>
mem_t<int> outer_join_count_matches(a_it a, int a_count, b_it b, int b_count,
                                     comp_t comp, context_t &context,
                                     int &append_count)
{
    mem_t<int> matches(b_count, context);
    mem_t<int> matches_count(1, context);
    // Compute lower and upper bounds of b into a.
    mem_t<int> lower_rev(b_count, context);
    mem_t<int> upper_rev(b_count, context);
    sorted_search<bounds_lower, launch_arg_t>(
        b, b_count, a, a_count, lower_rev.data(), comp, context
    );
    sorted_search<bounds_upper, launch_arg_t>(
        b, b_count, a, a_count, upper_rev.data(), comp, context
    );

    const int* lower_rev_data = lower_rev.data();
    const int* upper_rev_data = upper_rev.data();
    transform_scan<int>([=]MGPU_DEVICE(int index){
        return upper_rev_data[index] == lower_rev_data[index];
    }, b_count, matches.data(), plus_t<int>(), matches_count.data(), context);

    // Prepare output
    append_count = from_mem(matches_count)[0];
    return matches;
}



template<typename launch_arg_t = empty_t,
         typename a_it, typename b_it, typename comp_t>
mem_t<int> inner_join(a_it a, int a_count, b_it b, int b_count,
                       comp_t comp, context_t& context)
{
    _join_bounds bounds = compute_join_bounds(a, a_count, b, b_count, comp, context);
    int join_count;
    mem_t<int> scanned_sizes = scan_join_bounds(bounds, a_count, b_count, context, true,
                                                join_count);
    mem_t<int> output = compute_joined_indices(bounds, scanned_sizes, a_count,
                                               join_count, context, true);
    return output;
}


template<typename launch_arg_t = empty_t,
         typename a_it, typename b_it, typename comp_t>
mem_t<int> left_join(a_it a, int a_count, b_it b, int b_count,
                      comp_t comp, context_t& context)
{
    _join_bounds bounds = compute_join_bounds(a, a_count, b, b_count, comp, context);
    int join_count;
    mem_t<int> scanned_sizes = scan_join_bounds(bounds, a_count, b_count, context, false,
                                                join_count);
    mem_t<int> output = compute_joined_indices(bounds, scanned_sizes, a_count,
                                               join_count, context, false, 0);
    return output;
}

template<typename launch_arg_t = empty_t,
  typename a_it, typename b_it, typename comp_t>
mem_t<int> outer_join(a_it a, int a_count, b_it b, int b_count,
                       comp_t comp, context_t& context)
{
    _join_bounds bounds = compute_join_bounds(a, a_count, b, b_count, comp,
                                              context);
    int join_count;
    mem_t<int> scanned_sizes = scan_join_bounds(bounds, a_count, b_count, context, false,
                                                join_count);
    int append_count;
    mem_t<int> matches = outer_join_count_matches(a, a_count, b, b_count,
                                                  comp, context, append_count );
    mem_t<int> output = compute_joined_indices(bounds, scanned_sizes, a_count,
                                               join_count, context, false, append_count);
    outer_join_append_right(output.data(), matches, append_count, join_count,
                            context);
    return output;
}


struct join_result_base {
    virtual ~join_result_base() {}
    virtual void* data() = 0;
    virtual size_t size() = 0;
};

template <typename T>
struct join_result : public join_result_base {
    standard_context_t context;
    mem_t<T> result;

    join_result() : context(false) {}
    virtual void* data() {
        return result.data();
    }
    virtual size_t size() {
        return result.size();
    }
};

gdf_join_result_type* cffi_wrap(join_result_base *obj) {
    return reinterpret_cast<gdf_join_result_type*>(obj);
}

join_result_base* cffi_unwrap(gdf_join_result_type* hdl) {
    return reinterpret_cast<join_result_base*>(hdl);
}

} // end anony namespace

gdf_error gdf_join_result_free(gdf_join_result_type *result) {
    delete cffi_unwrap(result);
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
}

void* gdf_join_result_data(gdf_join_result_type *result) {
    return cffi_unwrap(result)->data();
}

size_t gdf_join_result_size(gdf_join_result_type *result) {
    return cffi_unwrap(result)->size();
}


// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
#define MAX_JOIN_SIZE (0xffffffffu)

#define DEF_JOIN(Fn, T, Joiner)                                             \
gdf_error gdf_##Fn(gdf_column *leftcol, gdf_column *rightcol,               \
                   gdf_join_result_type **out_result) {                     \
    using namespace mgpu;                                                   \
    if ( leftcol->dtype != rightcol->dtype) return GDF_UNSUPPORTED_DTYPE;   \
    if ( leftcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;   \
    if ( rightcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;  \
    std::unique_ptr<join_result<int> > result_ptr(new join_result<int>);    \
    result_ptr->result = Joiner((T*)leftcol->data, leftcol->size,           \
                                (T*)rightcol->data, rightcol->size,         \
                                less_t<T>(), result_ptr->context);          \
    CUDA_CHECK_LAST();                                                      \
    *out_result = cffi_wrap(result_ptr.release());                          \
    return GDF_SUCCESS;                                                     \
}


#define DEF_JOIN_DISP(Fn)                                                   \
gdf_error gdf_##Fn##_generic(gdf_column *leftcol, gdf_column * rightcol,    \
                                 gdf_join_result_type **out_result) {       \
    switch ( leftcol->dtype ){                                              \
    case GDF_INT8:  return gdf_##Fn##_i8(leftcol, rightcol, out_result);    \
    case GDF_INT32: return gdf_##Fn##_i32(leftcol, rightcol, out_result);   \
    case GDF_INT64: return gdf_##Fn##_i64(leftcol, rightcol, out_result);   \
    case GDF_FLOAT32: return gdf_##Fn##_f32(leftcol, rightcol, out_result); \
    case GDF_FLOAT64: return gdf_##Fn##_f64(leftcol, rightcol, out_result); \
    default: return GDF_UNSUPPORTED_DTYPE;                                  \
    }                                                                       \
}


#define DEF_INNER_JOIN(Fn, T) DEF_JOIN(inner_join_ ## Fn, T, inner_join)
DEF_JOIN_DISP(inner_join)
DEF_INNER_JOIN(i8,  int8_t)
DEF_INNER_JOIN(i32, int32_t)
DEF_INNER_JOIN(i64, int64_t)
DEF_INNER_JOIN(f32, float)
DEF_INNER_JOIN(f64, double)


#define DEF_LEFT_JOIN(Fn, T) DEF_JOIN(left_join_ ## Fn, T, left_join)
DEF_JOIN_DISP(left_join)
DEF_LEFT_JOIN(i8,  int8_t)
DEF_LEFT_JOIN(i32, int32_t)
DEF_LEFT_JOIN(i64, int64_t)
DEF_LEFT_JOIN(f32, float)
DEF_LEFT_JOIN(f64, double)


#define DEF_OUTER_JOIN(Fn, T) DEF_JOIN(outer_join_ ## Fn, T, outer_join)
DEF_JOIN_DISP(outer_join)
DEF_OUTER_JOIN(i8,  int8_t)
DEF_OUTER_JOIN(i32, int32_t)
DEF_OUTER_JOIN(i64, int64_t)
DEF_OUTER_JOIN(f32, float)
DEF_OUTER_JOIN(f64, double)

