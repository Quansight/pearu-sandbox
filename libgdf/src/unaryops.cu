#include "hip/hip_runtime.h"
#include <cmath>
#include <algorithm>

#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>

template<typename T, typename Tout, typename F>
__global__
void gpu_unary_op(const T *data, const gdf_valid_type *valid,
                  gdf_size_type size, Tout *results, F functor) {
    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;

    int start = tid + blkid * blksz;
    int step = blksz * gridsz;
    if ( valid ) {  // has valid mask
        for (int i=start; i<size; i+=step) {
            if ( gdf_is_valid(valid, i) )
                results[i] = functor.apply(data[i]);
        }
    } else {        // no valid mask
        for (int i=start; i<size; i+=step) {
            results[i] = functor.apply(data[i]);
        }
    }
}

template<typename T, typename Tout, typename F>
struct UnaryOp {
    static
    gdf_error launch(gdf_column *input, gdf_column *output) {
        /* check for size of the columns */
        if (input->size != output->size) {
            return GDF_COLUMN_SIZE_MISMATCH;
        }

        // find optimal blocksize
        int mingridsize, blocksize;
        CUDA_TRY(
            hipOccupancyMaxPotentialBlockSize(&mingridsize, &blocksize,
                                               gpu_unary_op<T, Tout, F>)
        );
        // find needed gridsize
        int neededgridsize = (input->size + blocksize - 1) / blocksize;
        int gridsize = std::min(neededgridsize, mingridsize);

        F functor;
        gpu_unary_op<<<gridsize, blocksize>>>(
            // input
            (const T*)input->data, input->valid, input->size,
            // output
            (Tout*)output->data,
            // action
            functor
        );

        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }
};


template<typename T, typename F>
struct MathOp {
    static
    gdf_error launch(gdf_column *input, gdf_column *output) {
        return UnaryOp<T, T, F>::launch(input, output);
    }
};


#define DEF_UNARY_OP_REAL(F)                                        \
gdf_error F##_generic(gdf_column *input, gdf_column *output) {      \
    switch ( input->dtype ) {                                       \
    case GDF_FLOAT32: return F##_f32(input, output);                \
    case GDF_FLOAT64: return F##_f64(input, output);                \
    default: return GDF_UNSUPPORTED_DTYPE;                          \
    }                                                               \
}

#define DEF_CAST_OP(TO)                                                       \
gdf_error gdf_cast_generic_to_##TO(gdf_column *input, gdf_column *output) {   \
    switch ( input->dtype ) {                                                 \
    case    GDF_INT8: return gdf_cast_i8_to_##TO(input, output);              \
    case   GDF_INT32: return gdf_cast_i32_to_##TO(input, output);             \
    case   GDF_INT64: return gdf_cast_i64_to_##TO(input, output);             \
    case GDF_FLOAT32: return gdf_cast_f32_to_##TO(input, output);             \
    case GDF_FLOAT64: return gdf_cast_f64_to_##TO(input, output);             \
    default: return GDF_UNSUPPORTED_DTYPE;                                    \
    }                                                                         \
}

// trig functions

template<typename T>
struct DeviceSin {
    __device__
    T apply(T data) {
        return std::sin(data);
    }
};

template<typename T>
struct DeviceCos {
    __device__
    T apply(T data) {
        return std::cos(data);
    }
};

template<typename T>
struct DeviceTan {
    __device__
    T apply(T data) {
        return std::tan(data);
    }
};

template<typename T>
struct DeviceArcSin {
    __device__
    T apply(T data) {
        return std::asin(data);
    }
};

template<typename T>
struct DeviceArcCos {
    __device__
    T apply(T data) {
        return std::acos(data);
    }
};

template<typename T>
struct DeviceArcTan {
    __device__
    T apply(T data) {
        return std::atan(data);
    }
};

DEF_UNARY_OP_REAL(gdf_sin)

gdf_error gdf_sin_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceSin<float> >::launch(input, output);
}

gdf_error gdf_sin_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceSin<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_cos)

gdf_error gdf_cos_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceCos<float> >::launch(input, output);
}

gdf_error gdf_cos_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceCos<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_tan)

gdf_error gdf_tan_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceTan<float> >::launch(input, output);
}

gdf_error gdf_tan_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceTan<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_asin)

gdf_error gdf_asin_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcSin<float> >::launch(input, output);
}

gdf_error gdf_asin_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcSin<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_acos)

gdf_error gdf_acos_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcCos<float> >::launch(input, output);
}

gdf_error gdf_acos_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcCos<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_atan)

gdf_error gdf_atan_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcTan<float> >::launch(input, output);
}

gdf_error gdf_atan_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcTan<double> >::launch(input, output);
}

// exponential functions

template<typename T>
struct DeviceExp {
    __device__
    T apply(T data) {
        return std::exp(data);
    }
};

template<typename T>
struct DeviceLog {
    __device__
    T apply(T data) {
        return std::log(data);
    }
};

DEF_UNARY_OP_REAL(gdf_exp)

gdf_error gdf_exp_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceExp<float> >::launch(input, output);
}

gdf_error gdf_exp_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceExp<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_log)

gdf_error gdf_log_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceLog<float> >::launch(input, output);
}

gdf_error gdf_log_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceLog<double> >::launch(input, output);
}

// exponential functions

template<typename T>
struct DeviceSqrt {
    __device__
    T apply(T data) {
        return std::sqrt(data);
    }
};

DEF_UNARY_OP_REAL(gdf_sqrt)

gdf_error gdf_sqrt_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceSqrt<float> >::launch(input, output);
}

gdf_error gdf_sqrt_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceSqrt<double> >::launch(input, output);
}

// rounding functions

template<typename T>
struct DeviceCeil {
    __device__
    T apply(T data) {
        return std::ceil(data);
    }
};

template<typename T>
struct DeviceFloor {
    __device__
    T apply(T data) {
        return std::floor(data);
    }
};

DEF_UNARY_OP_REAL(gdf_ceil)

gdf_error gdf_ceil_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceCeil<float> >::launch(input, output);
}

gdf_error gdf_ceil_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceCeil<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_floor)

gdf_error gdf_floor_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceFloor<float> >::launch(input, output);
}

gdf_error gdf_floor_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceFloor<double> >::launch(input, output);
}


// casting

template<typename From, typename To>
struct DeviceCast {
    __device__
    To apply(From data) {
        return (To)data;
    }
};

#define DEF_CAST_IMPL(VFROM, VTO, TFROM, TTO)                                 \
gdf_error gdf_cast_##VFROM##_to_##VTO(gdf_column *input, gdf_column *output)  \
{ return UnaryOp<TFROM, TTO, DeviceCast<TFROM, TTO> >::launch(input, output); }


DEF_CAST_OP(f32)
DEF_CAST_IMPL(i8,  f32,  int8_t, float)
DEF_CAST_IMPL(i32, f32, int32_t, float)
DEF_CAST_IMPL(i64, f32, int64_t, float)
DEF_CAST_IMPL(f32, f32,   float, float)
DEF_CAST_IMPL(f64, f32,  double, float)

DEF_CAST_OP(f64)
DEF_CAST_IMPL(i8,  f64,  int8_t, double)
DEF_CAST_IMPL(i32, f64, int32_t, double)
DEF_CAST_IMPL(i64, f64, int64_t, double)
DEF_CAST_IMPL(f32, f64,   float, double)
DEF_CAST_IMPL(f64, f64,  double, double)

DEF_CAST_OP(i8)
DEF_CAST_IMPL(i8,  i8,  int8_t, int8_t)
DEF_CAST_IMPL(i32, i8, int32_t, int8_t)
DEF_CAST_IMPL(i64, i8, int64_t, int8_t)
DEF_CAST_IMPL(f32, i8,   float, int8_t)
DEF_CAST_IMPL(f64, i8,  double, int8_t)

DEF_CAST_OP(i32)
DEF_CAST_IMPL(i8,  i32,  int8_t, int32_t)
DEF_CAST_IMPL(i32, i32, int32_t, int32_t)
DEF_CAST_IMPL(i64, i32, int64_t, int32_t)
DEF_CAST_IMPL(f32, i32,   float, int32_t)
DEF_CAST_IMPL(f64, i32,  double, int32_t)

DEF_CAST_OP(i64)
DEF_CAST_IMPL(i8,  i64,  int8_t, int64_t)
DEF_CAST_IMPL(i32, i64, int32_t, int64_t)
DEF_CAST_IMPL(i64, i64, int64_t, int64_t)
DEF_CAST_IMPL(f32, i64,   float, int64_t)
DEF_CAST_IMPL(f64, i64,  double, int64_t)
