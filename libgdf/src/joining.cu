/*
 * Copyright (c) 2017, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gdf/gdf.h>
#include <gdf/errorutils.h>

#include <memory>
#include <iostream>

#include "joining.h"

using namespace mgpu;

template <typename T>
void dump_mem(const char name[], const mem_t<T> & mem) {

    auto data = from_mem(mem);
    std::cout << name << " = " ;
    for (int i=0; i < data.size(); ++i) {
        std::cout << data[i] << ", ";
    }
    std::cout << "\n";
}

gdf_join_result_type* cffi_wrap(join_result_base *obj) {
    return reinterpret_cast<gdf_join_result_type*>(obj);
}

join_result_base* cffi_unwrap(gdf_join_result_type* hdl) {
    return reinterpret_cast<join_result_base*>(hdl);
}

gdf_error gdf_join_result_free(gdf_join_result_type *result) {
    delete cffi_unwrap(result);
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
}

void* gdf_join_result_data(gdf_join_result_type *result) {
    return cffi_unwrap(result)->data();
}

size_t gdf_join_result_size(gdf_join_result_type *result) {
    return cffi_unwrap(result)->size();
}


// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
#define MAX_JOIN_SIZE (0xffffffffu)

#define DEF_JOIN(Fn, T, Joiner)                                             \
gdf_error gdf_##Fn(gdf_column *leftcol, gdf_column *rightcol,               \
                   gdf_join_result_type **out_result) {                     \
    using namespace mgpu;                                                   \
    if ( leftcol->dtype != rightcol->dtype) return GDF_UNSUPPORTED_DTYPE;   \
    if ( leftcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;   \
    if ( rightcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;  \
    std::unique_ptr<join_result<int> > result_ptr(new join_result<int>);    \
    result_ptr->result = Joiner((T*)leftcol->data, leftcol->size,           \
                                (T*)rightcol->data, rightcol->size,         \
                                less_t<T>(), result_ptr->context);          \
    CUDA_CHECK_LAST();                                                      \
    *out_result = cffi_wrap(result_ptr.release());                          \
    return GDF_SUCCESS;                                                     \
}

//TODO: DEF_JOIN_HASH can be merged with DEF_JOIN conce inner_join is using gdf_size_type
#define DEF_JOIN_HASH(Fn, T, Joiner, JoinType)                              \
gdf_error gdf_##Fn(gdf_column *leftcol, gdf_column *rightcol,               \
                   gdf_join_result_type **out_result) {                     \
    using namespace mgpu;                                                   \
    if ( leftcol->dtype != rightcol->dtype) return GDF_UNSUPPORTED_DTYPE;   \
    if ( leftcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;   \
    if ( rightcol->size >= MAX_JOIN_SIZE ) return GDF_COLUMN_SIZE_TOO_BIG;  \
    std::unique_ptr<join_result<int> > result_ptr(new join_result<int>);    \
    result_ptr->result = Joiner<JoinType>((T*)leftcol->data, (int)leftcol->size,      \
                                (T*)rightcol->data, (int)rightcol->size,    \
				(int32_t*)NULL, (int32_t*)NULL,		    \
				(int32_t*)NULL, (int32_t*)NULL,		    \
                                less_t<T>(), result_ptr->context);          \
    CUDA_CHECK_LAST();                                                      \
    *out_result = cffi_wrap(result_ptr.release());                          \
    return GDF_SUCCESS;                                                     \
}

#define DEF_JOIN_DISP(Fn)                                                   \
gdf_error gdf_##Fn##_generic(gdf_column *leftcol, gdf_column * rightcol,    \
                                 gdf_join_result_type **out_result) {       \
    switch ( leftcol->dtype ){                                              \
    case GDF_INT8:  return gdf_##Fn##_i8(leftcol, rightcol, out_result);    \
    case GDF_INT32: return gdf_##Fn##_i32(leftcol, rightcol, out_result);   \
    case GDF_INT64: return gdf_##Fn##_i64(leftcol, rightcol, out_result);   \
    case GDF_FLOAT32: return gdf_##Fn##_f32(leftcol, rightcol, out_result); \
    case GDF_FLOAT64: return gdf_##Fn##_f64(leftcol, rightcol, out_result); \
    default: return GDF_UNSUPPORTED_DTYPE;                                  \
    }                                                                       \
}

#define JOIN_HASH_TYPES(T1, l1, r1, T2, l2, r2, T3, l3, r3) \
  result_ptr->result = join_hash<LEFT_JOIN>( \
				(T1*)l1, (int)leftcol[0]->size, \
                                (T1*)r1, (int)rightcol[0]->size, \
                                (T2*)l2, (T2*)r2, \
                                (T3*)l3, (T3*)r3, \
                                less_t<int64_t>(), result_ptr->context);

#define JOIN_HASH_T3(T1, l1, r1, T2, l2, r2, T3, l3, r3) \
  if (T3 == GDF_INT8)  { JOIN_HASH_TYPES(T1, l1, r1, T2, l2, r2, int8_t, l3, r3) } \
  if (T3 == GDF_INT16) { JOIN_HASH_TYPES(T1, l1, r1, T2, l2, r2, int16_t, l3, r3) } \
  if (T3 == GDF_INT32) { JOIN_HASH_TYPES(T1, l1, r1, T2, l2, r2, int32_t, l3, r3) } \
  if (T3 == GDF_INT64) { JOIN_HASH_TYPES(T1, l1, r1, T2, l2, r2, int64_t, l3, r3) }

#define JOIN_HASH_T2(T1, l1, r1, T2, l2, r2, T3, l3, r3) \
  if (T2 == GDF_INT8)  { JOIN_HASH_T3(T1, l1, r1, int8_t, l2, r2, T3, l3, r3) } \
  if (T2 == GDF_INT16) { JOIN_HASH_T3(T1, l1, r1, int16_t, l2, r2, T3, l3, r3) } \
  if (T2 == GDF_INT32) { JOIN_HASH_T3(T1, l1, r1, int32_t, l2, r2, T3, l3, r3) } \
  if (T2 == GDF_INT64) { JOIN_HASH_T3(T1, l1, r1, int64_t, l2, r2, T3, l3, r3) }

#define JOIN_HASH_T1(T1, l1, r1, T2, l2, r2, T3, l3, r3) \
  if (T1 == GDF_INT8)  { JOIN_HASH_T2(int8_t, l1, r1, T2, l2, r2, T3, l3, r3) } \
  if (T1 == GDF_INT16) { JOIN_HASH_T2(int16_t, l1, r1, T2, l2, r2, T3, l3, r3) } \
  if (T1 == GDF_INT32) { JOIN_HASH_T2(int32_t, l1, r1, T2, l2, r2, T3, l3, r3) } \
  if (T1 == GDF_INT64) { JOIN_HASH_T2(int64_t, l1, r1, T2, l2, r2, T3, l3, r3) }

// multi-column join function
gdf_error gdf_multi_left_join_generic(int num_cols, gdf_column **leftcol, gdf_column **rightcol, gdf_join_result_type **out_result)
{
  // check that the columns have matching types and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    if (rightcol[i]->dtype != leftcol[i]->dtype) return GDF_JOIN_DTYPE_MISMATCH;
    if (i > 0 && leftcol[i]->size != leftcol[i-1]->size) return GDF_COLUMN_SIZE_MISMATCH;
    if (i > 0 && rightcol[i]->size != rightcol[i-1]->size) return GDF_COLUMN_SIZE_MISMATCH;
  }

  // TODO: currently support up to 3 columns, and only int32 and int64 types
  if (num_cols > 3) return GDF_JOIN_TOO_MANY_COLUMNS;
  for (int i = 0; i < num_cols; i++) {
    if (leftcol[i]->dtype != GDF_INT8 &&
        leftcol[i]->dtype != GDF_INT16 &&
        leftcol[i]->dtype != GDF_INT32 &&
	leftcol[i]->dtype != GDF_INT64) return GDF_UNSUPPORTED_DTYPE;
  }

  std::unique_ptr<join_result<int> > result_ptr(new join_result<int>);
  switch (num_cols) {
  case 1:
    JOIN_HASH_T1(leftcol[0]->dtype, leftcol[0]->data, rightcol[0]->data,
		 GDF_INT32, NULL, NULL,
		 GDF_INT32, NULL, NULL)
    break;
  case 2:
    JOIN_HASH_T1(leftcol[0]->dtype, leftcol[0]->data, rightcol[0]->data,
		 leftcol[1]->dtype, leftcol[1]->data, rightcol[1]->data,
		 GDF_INT32, NULL, NULL)
    break;
  case 3:
    JOIN_HASH_T1(leftcol[0]->dtype, leftcol[0]->data, rightcol[0]->data,
		 leftcol[1]->dtype, leftcol[1]->data, rightcol[1]->data,
		 leftcol[2]->dtype, leftcol[2]->data, rightcol[2]->data)
    break;
  }

  CUDA_CHECK_LAST();
  *out_result = cffi_wrap(result_ptr.release());
  return GDF_SUCCESS;
}

#ifdef HASH_JOIN
#define DEF_INNER_JOIN(Fn, T) DEF_JOIN_HASH(inner_join_ ## Fn, T, join_hash, INNER_JOIN)
#define DEF_INNER_JOIN_FP(Fn, T) DEF_JOIN(inner_join_ ## Fn, T, inner_join)
#else
#define DEF_INNER_JOIN(Fn, T) DEF_JOIN(inner_join_ ## Fn, T, inner_join)
#define DEF_INNER_JOIN_FP(Fn, T) DEF_JOIN(inner_join_ ## Fn, T, inner_join)
#endif
DEF_JOIN_DISP(inner_join)
DEF_INNER_JOIN(i8,  int8_t)
DEF_INNER_JOIN(i16, int16_t)
DEF_INNER_JOIN(i32, int32_t)
DEF_INNER_JOIN(i64, int64_t)
DEF_INNER_JOIN_FP(f32, float)
DEF_INNER_JOIN_FP(f64, double)


#ifdef HASH_JOIN
#define DEF_LEFT_JOIN(Fn, T) DEF_JOIN_HASH(left_join_ ## Fn, T, join_hash, LEFT_JOIN)
#define DEF_LEFT_JOIN_FP(Fn, T) DEF_JOIN(left_join_ ## Fn, T, left_join)
#else
#define DEF_LEFT_JOIN(Fn, T) DEF_JOIN(left_join_ ## Fn, T, left_join)
#define DEF_LEFT_JOIN_FP(Fn, T) DEF_JOIN(left_join_ ## Fn, T, left_join)
#endif
DEF_JOIN_DISP(left_join)
DEF_LEFT_JOIN(i8,  int8_t)
DEF_LEFT_JOIN(i32, int32_t)
DEF_LEFT_JOIN(i64, int64_t)
DEF_LEFT_JOIN_FP(f32, float)
DEF_LEFT_JOIN_FP(f64, double)


#define DEF_OUTER_JOIN(Fn, T) DEF_JOIN(outer_join_ ## Fn, T, outer_join)
DEF_JOIN_DISP(outer_join)
DEF_OUTER_JOIN(i8,  int8_t)
DEF_OUTER_JOIN(i32, int32_t)
DEF_OUTER_JOIN(i64, int64_t)
DEF_OUTER_JOIN(f32, float)
DEF_OUTER_JOIN(f64, double)

